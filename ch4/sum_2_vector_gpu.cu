#include "hip/hip_runtime.h"
#include "../common/book.h"
#include <time.h>
#define N 100

__global__ void add_block(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

__global__ void add_thread(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}


int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeof(int)));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; ++i) {
        a[i] = -i;
        b[i] = i * i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDEL_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add_block<<<N, 1>>>(dev_a, dev_b, dev_c);

    add_thread<<<1, N>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
