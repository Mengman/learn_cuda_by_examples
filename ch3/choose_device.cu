#include "../common/book.h"

int main(void) {
  hipDeviceProp_t prop;
  int dev;

  HANDLE_ERROR(hipGetDevice(&dev));
  printf("ID of current CUDA device: %d\n", dev);

  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 7;
  prop.minor = 5;

  HANDLE_ERROR(hipChooseDevice(&dev, &prop));
  printf("ID of CUDA device closest to revision %d.%d: %d\n", prop.major,
         prop.minor, dev);
  HANDLE_ERROR(hipSetDevice(dev));
}